#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/upsample_layer.hpp"

namespace caffe {

template <typename Dtype>
  __global__ void UpsampleForward(const int nthreads, int in_w, int in_h,
      int out_w, int out_h, const Dtype* bottom_data,
      const Dtype* upsample_index_data, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int c = index / (in_w * in_h);
      int i = index - c * in_w * in_h;
      int j = c * out_w * out_h;
      int upsample_idx = static_cast<int>(upsample_index_data[i]);
      top_data[j + upsample_idx] = bottom_data[index];
    }
  }

template <typename Dtype>
void UpsampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* upsample_index_data = upsample_index_.gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_set(top[0]->count(), Dtype(0), top_data);
  int bottom_count = bottom[0]->count();
  UpsampleForward<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
      bottom_count, width_, height_, upsample_w_, upsample_h_,
      bottom_data, upsample_index_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
  __global__ void UpsampleBackward(const int nthreads, int in_w, int in_h,
      int out_w, int out_h, const Dtype* top_diff,
      const Dtype* upsample_index_data, Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int c = index / (in_w * in_h);
      int i = index - c * in_w * in_h;
      int j = c * out_w * out_h;
      int upsample_idx = static_cast<int>(upsample_index_data[i]);
      bottom_diff[index] = top_diff[j + upsample_idx];
    }
  }

template <typename Dtype>
void UpsampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* upsample_index_data = upsample_index_.gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    caffe_gpu_set(bottom_count, Dtype(0.), bottom_diff);
    UpsampleBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom_count, width_, height_, upsample_w_, upsample_h_,
        top_diff, upsample_index_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(UpsampleLayer);


}  // namespace caffe
