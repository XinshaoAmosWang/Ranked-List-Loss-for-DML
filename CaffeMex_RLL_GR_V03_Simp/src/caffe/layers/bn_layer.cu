#include <algorithm>
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/bn_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

	template <typename Dtype>
	void BNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* const_bottom_data = bottom[0]->gpu_data();
		const Dtype* const_top_data = top[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();

		const Dtype* scale_data = this->blobs_[0]->gpu_data();
		const Dtype* shift_data = this->blobs_[1]->gpu_data();

		// ---------- mean subtraction ---------- //
      // statistic across spatial
      caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_, Dtype(1. / (height_ * width_)), const_bottom_data,
                            spatial_sum_multiplier_.gpu_data(), Dtype(0), spatial_statistic_.mutable_gpu_data());
      // statistic across batch
      caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1. / num_), spatial_statistic_.gpu_data(),
		  batch_sum_multiplier_.gpu_data(), Dtype(0), ex_.mutable_gpu_data());
		if (this->phase_ == TRAIN) {
			// sync statistics
			if ( sync_forward_ ){
				// first, sync EX
				caffe_copy(channels_, ex_.gpu_data(), statistics_all_.mutable_gpu_data());
				P2PSync<Dtype>* p2p = this->callbacks()[ 0 ]->callbacks()[ 0 ]->p2p()[ 0 ];
				Blob<Dtype> statistics_child(1, channels_, 1, 1);
				for ( int i = 0; i < p2p->children().size(); ++i ){
#ifdef _WIN64
					Blob<Dtype>* s_c_ogpu = p2p->dataQueue().pop();
#else
					Blob<Dtype>* s_c_ogpu = NULL;
					while ( !p2p->dataQueue().try_pop(&s_c_ogpu) )
						;
#endif
					//Blob<Dtype>* s_c_ogpu = p2p->dataQueue().pop();
					CUDA_CHECK(hipMemcpyAsync(statistics_child.mutable_gpu_data(), s_c_ogpu->gpu_data(), channels_*sizeof( Dtype ), hipMemcpyDeviceToDevice, hipStreamDefault));
					CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
					caffe_gpu_add(channels_, statistics_child.gpu_data(), statistics_all_.gpu_data(), statistics_all_.mutable_gpu_data());
				}
				if ( p2p->parent() ){
					p2p->parent()->dataQueue().push(&statistics_all_);
#ifdef _WIN64
					Blob<Dtype>* statistics_final = p2p->dataQueue().pop();
#else
					Blob<Dtype>* statistics_final = NULL;
					while ( !p2p->dataQueue().try_pop(&statistics_final) )
						;
#endif
					//Blob<Dtype>* statistics_final = p2p->dataQueue().pop();
					CUDA_CHECK(hipMemcpyAsync(ex_.mutable_gpu_data(), statistics_final->gpu_data(), channels_*sizeof( Dtype ), hipMemcpyDeviceToDevice, hipStreamDefault));
					CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
				}
				else {
					caffe_gpu_scal<Dtype>(channels_, Dtype(1.0 / Caffe::solver_count()), statistics_all_.mutable_gpu_data());
					caffe_copy(channels_, statistics_all_.gpu_data(), ex_.mutable_gpu_data());
				}
				for ( int i = 0; i < p2p->children().size(); ++i ){
					p2p->children()[ i ]->dataQueue().push(&ex_);
				}
			}
			// save history mean
			caffe_gpu_axpby(ex_.count(), Dtype(1) - decay_, ex_.gpu_data(), decay_,
			    this->blobs_[2]->mutable_gpu_data());
		}
		if (this->phase_ == TEST && moving_average_) {
			// use moving average mean
			caffe_copy(ex_.count(), this->blobs_[ 2 ]->gpu_data(), ex_.mutable_gpu_data());
		}
		
		// put mean blob into buffer_blob_
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
			batch_sum_multiplier_.gpu_data(), ex_.gpu_data(), Dtype(0),
			spatial_statistic_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(-1),
			spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(0),
			buffer_blob_.mutable_gpu_data());
		// substract mean
		caffe_gpu_add(buffer_blob_.count(), const_bottom_data, buffer_blob_.gpu_data(), top_data);

		// ---------- variance normalization ---------- //
		// add by yu liu
		// calculate EX2
		caffe_gpu_powx(bottom[ 0 ]->count(), const_bottom_data, Dtype(2), buffer_blob_.mutable_gpu_data());
		// statistic across spatial
		caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_, Dtype(1. / ( height_ * width_ )), buffer_blob_.gpu_data(),
			spatial_sum_multiplier_.gpu_data(), Dtype(0), spatial_statistic_.mutable_gpu_data());
		// statistic across batch
		caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1. / num_), spatial_statistic_.gpu_data(),
			batch_sum_multiplier_.gpu_data(), Dtype(0), batch_statistic_.mutable_gpu_data());
		

		// original dx
      //// put the squares of X - mean into buffer_blob_
      //caffe_gpu_powx(buffer_blob_.count(), const_top_data, Dtype(2), buffer_blob_.mutable_gpu_data());
      //// statistic across spatial
      //caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_, Dtype(1. / (height_ * width_)), buffer_blob_.gpu_data(),
      //                      spatial_sum_multiplier_.gpu_data(), Dtype(0), spatial_statistic_.mutable_gpu_data());
      //// statistic across batch
      //caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1. / num_), spatial_statistic_.gpu_data(),
      //                      batch_sum_multiplier_.gpu_data(), Dtype(0), batch_statistic_.mutable_gpu_data());
    if (this->phase_ == TRAIN) {
		if ( sync_forward_ ){
			// second, sync EX2
			caffe_copy(channels_, batch_statistic_.gpu_data(), statistics_all_.mutable_gpu_data());
			P2PSync<Dtype>* p2p = this->callbacks()[ 0 ]->callbacks()[ 0 ]->p2p()[ 0 ];
			Blob<Dtype> statistics_child(1, channels_, 1, 1);
			for ( int i = 0; i < p2p->children().size(); ++i ){
#ifdef _WIN64
				Blob<Dtype>* s_c_ogpu = p2p->dataQueue().pop();
#else // Linux support
				Blob<Dtype>* s_c_ogpu = NULL;
				while ( !p2p->dataQueue().try_pop(&s_c_ogpu) )
					;
#endif
				//Blob<Dtype>* s_c_ogpu = p2p->dataQueue().pop();
				CUDA_CHECK(hipMemcpyAsync(statistics_child.mutable_gpu_data(), s_c_ogpu->gpu_data(), channels_*sizeof( Dtype ), hipMemcpyDeviceToDevice, hipStreamDefault));
				CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
				caffe_gpu_add(channels_, statistics_child.gpu_data(), statistics_all_.gpu_data(), statistics_all_.mutable_gpu_data());
			}
			if ( p2p->parent() ){
				p2p->parent()->dataQueue().push(&statistics_all_);
#ifdef _WIN64
				Blob<Dtype>* statistics_final = p2p->dataQueue().pop();
#else // Linux support
				Blob<Dtype>* statistics_final = NULL;
				while ( !p2p->dataQueue().try_pop(&statistics_final) )
					;
#endif
				//Blob<Dtype>* statistics_final = p2p->dataQueue().pop();
				CUDA_CHECK(hipMemcpyAsync(dx_.mutable_gpu_data(), statistics_final->gpu_data(), channels_*sizeof( Dtype ), hipMemcpyDeviceToDevice, hipStreamDefault));
				CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
			}
			else {
				caffe_gpu_scal<Dtype>(channels_, Dtype(1.0 / Caffe::solver_count()), statistics_all_.mutable_gpu_data());
				Blob<Dtype> e2x_(1, channels_, 1, 1);
				caffe_gpu_powx(ex_.count(), ex_.gpu_data(), Dtype(2), e2x_.mutable_gpu_data());
				caffe_gpu_sub<Dtype>(ex_.count(), statistics_all_.gpu_data(), e2x_.gpu_data(), dx_.mutable_gpu_data());
			}
			for ( int i = 0; i < p2p->children().size(); ++i ){
				p2p->children()[ i ]->dataQueue().push(&dx_);
			}
		}
		else{
			Blob<Dtype> e2x_(1, channels_, 1, 1);
			caffe_gpu_powx(ex_.count(), ex_.gpu_data(), Dtype(2), e2x_.mutable_gpu_data());
			caffe_gpu_sub<Dtype>(ex_.count(), batch_statistic_.gpu_data(), e2x_.gpu_data(), dx_.mutable_gpu_data());
		}


      // save history variance
		caffe_gpu_axpby(dx_.count(), Dtype(1) - decay_, dx_.gpu_data(), decay_,
                      this->blobs_[3]->mutable_gpu_data());
    }
	if (this->phase_ == TEST ) {
		// use moving average variance
		if ( moving_average_ )
			caffe_copy(dx_.count(), this->blobs_[ 3 ]->gpu_data(), dx_.mutable_gpu_data());
		else{
			Blob<Dtype> e2x_(1, channels_, 1, 1);
			caffe_gpu_powx(ex_.count(), ex_.gpu_data(), Dtype(2), e2x_.mutable_gpu_data());
			caffe_gpu_sub<Dtype>(ex_.count(), batch_statistic_.gpu_data(), e2x_.gpu_data(), dx_.mutable_gpu_data());
		}
	}
    
	// Until now, dx_ should be calculated

    // add eps
	caffe_gpu_add_scalar(dx_.count(), var_eps_, dx_.mutable_gpu_data());
		// std
	caffe_gpu_powx(dx_.count(), dx_.gpu_data(), Dtype(0.5),
			batch_statistic_.mutable_gpu_data());
		// put std blob into buffer_blob_
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
			batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(), Dtype(0),
			spatial_statistic_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(1),
			spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(0),
			buffer_blob_.mutable_gpu_data());
		// variance normalization
		caffe_gpu_div(buffer_blob_.count(), const_top_data, buffer_blob_.gpu_data(), top_data);

		// ---------- save x_norm and x_std ---------- //
		caffe_copy(buffer_blob_.count(), const_top_data, x_norm_.mutable_gpu_data());
		caffe_copy(batch_statistic_.count(), batch_statistic_.gpu_data(), x_std_.mutable_gpu_data());

		// ---------- scale ---------- //
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
			batch_sum_multiplier_.gpu_data(), scale_data, Dtype(0),
			spatial_statistic_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(1),
			spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(0),
			buffer_blob_.mutable_gpu_data());
		caffe_gpu_mul(buffer_blob_.count(), const_top_data, buffer_blob_.gpu_data(), top_data);

		// ---------- shift ---------- //
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
			batch_sum_multiplier_.gpu_data(), shift_data, Dtype(0),
			spatial_statistic_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(1),
			spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(0),
			buffer_blob_.mutable_gpu_data());
		caffe_gpu_add(buffer_blob_.count(), const_top_data, buffer_blob_.gpu_data(), top_data);

	}

	template <typename Dtype>
	void BNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		const Dtype* const_bottom_diff = bottom[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		const Dtype* const_top_diff = top[0]->gpu_diff();	

		Dtype* scale_diff = this->blobs_[0]->mutable_gpu_diff();
		Dtype* shift_diff = this->blobs_[1]->mutable_gpu_diff();
		const Dtype* scale_data = this->blobs_[0]->gpu_data();

		// ---------- gradient w.r.t. scale ---------- //
		caffe_gpu_mul(buffer_blob_.count(), x_norm_.gpu_data(), const_top_diff, buffer_blob_.mutable_gpu_data());
		// statistic across spatial
		caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_, Dtype(1), buffer_blob_.gpu_data(),
			spatial_sum_multiplier_.gpu_data(), Dtype(0), spatial_statistic_.mutable_gpu_data());
		// statistic across batch
		caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1), spatial_statistic_.gpu_data(),
			batch_sum_multiplier_.gpu_data(), Dtype(0), scale_diff);

		// ---------- gradient w.r.t. shift ---------- //
		// statistic across spatial
		caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_, Dtype(1), const_top_diff,
			spatial_sum_multiplier_.gpu_data(), Dtype(0), spatial_statistic_.mutable_gpu_data());
		// statistic across batch
		caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1), spatial_statistic_.gpu_data(),
			batch_sum_multiplier_.gpu_data(), Dtype(0), shift_diff);

		// ---------- gradient w.r.t. to bottom blob ---------- //
		// put scale * top_diff to buffer_blob_
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
			batch_sum_multiplier_.gpu_data(), scale_data, Dtype(0),
			spatial_statistic_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(1),
			spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(0),
			buffer_blob_.mutable_gpu_data());
		caffe_gpu_mul(buffer_blob_.count(), const_top_diff, buffer_blob_.gpu_data(), buffer_blob_.mutable_gpu_data());

    if (this->phase_ == TRAIN) {
      // use new top diff for computation
      caffe_gpu_mul(buffer_blob_.count(), x_norm_.gpu_data(), buffer_blob_.gpu_data(), bottom_diff);
      // statistic across spatial
      caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_, Dtype(1), const_bottom_diff,
                            spatial_sum_multiplier_.gpu_data(), Dtype(0), spatial_statistic_.mutable_gpu_data());
      // statistic across batch
      caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1), spatial_statistic_.gpu_data(),
                            batch_sum_multiplier_.gpu_data(), Dtype(0), batch_statistic_.mutable_gpu_data());

      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
                            batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(), Dtype(0),
                            spatial_statistic_.mutable_gpu_data());
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(1),
                            spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(0),
                            bottom_diff);

      caffe_gpu_mul(buffer_blob_.count(), x_norm_.gpu_data(), const_bottom_diff, bottom_diff);

      // statistic across spatial
      caffe_gpu_gemv<Dtype>(CblasNoTrans, num_ * channels_, height_ * width_, Dtype(1), buffer_blob_.gpu_data(),
                            spatial_sum_multiplier_.gpu_data(), Dtype(0), spatial_statistic_.mutable_gpu_data());
      // statistic across batch
      caffe_gpu_gemv<Dtype>(CblasTrans, num_, channels_, Dtype(1), spatial_statistic_.gpu_data(),
                            batch_sum_multiplier_.gpu_data(), Dtype(0), batch_statistic_.mutable_gpu_data());

      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
                            batch_sum_multiplier_.gpu_data(), batch_statistic_.gpu_data(), Dtype(0),
                            spatial_statistic_.mutable_gpu_data());
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(1),
                            spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(1),
                            bottom_diff);

      caffe_gpu_axpby(buffer_blob_.count(), Dtype(1), buffer_blob_.gpu_data(), Dtype(-1. / (num_ * height_ * width_)),
                      bottom_diff);
    }
    if (this->phase_ == TEST && moving_average_) {
      // use moving average variance
      caffe_copy(buffer_blob_.count(), buffer_blob_.gpu_data(), bottom_diff);
    }
        
		// variance normalization
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_, channels_, 1, Dtype(1),
			batch_sum_multiplier_.gpu_data(), x_std_.gpu_data(), Dtype(0),
			spatial_statistic_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_ * channels_, height_ * width_, 1, Dtype(1),
			spatial_statistic_.gpu_data(), spatial_sum_multiplier_.gpu_data(), Dtype(0),
			buffer_blob_.mutable_gpu_data());

		caffe_gpu_div(buffer_blob_.count(), const_bottom_diff, buffer_blob_.gpu_data(), bottom_diff);

	}

	INSTANTIATE_LAYER_GPU_FUNCS(BNLayer);

}  // namespace caffe
