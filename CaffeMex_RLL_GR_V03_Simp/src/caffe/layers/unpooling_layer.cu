#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/unpooling_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxUnpoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int unpooled_height, const int unpooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* top_data,
    const Dtype* bottom_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % width;
    int ph = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
  
    int uph = max(0, min(ph * stride_h - pad_h, unpooled_height - 1));
    int upw = max(0, min(pw * stride_w - pad_w, unpooled_width - 1));
    int unpooled_index = uph * unpooled_width + upw;

    top_data += (n * channels + c) * unpooled_height * unpooled_width;
    if (bottom_mask) {
      const int mask_index = bottom_mask[index];
      top_data[mask_index] = bottom_data[index]; 
    } else {
      top_data[unpooled_index] = bottom_data[index];
    } 
  }
}

template <typename Dtype>
__global__ void AveUnpoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int unpooled_height,
    const int unpooled_width, const int height, const int width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % unpooled_width + pad_w;
    int h = (index / unpooled_width) % unpooled_height + pad_h;
    int c = (index / unpooled_width / unpooled_height) % channels;
    int n = index / unpooled_width / unpooled_height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, width);
    Dtype distval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, unpooled_height + pad_h);
        int wend = min(wstart + kernel_w, unpooled_width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        distval += bottom_data[ph * width + pw] / pool_size;
      }
    }
    top_data[index] = distval;
  }
}

template <typename Dtype>
__global__ void TileUnpoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int unpooled_height,
    const int unpooled_width, const int height, const int width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % unpooled_width + pad_w;
    int h = (index / unpooled_width) % unpooled_height + pad_h;
    int c = (index / unpooled_width / unpooled_height) % channels;
    int n = index / unpooled_width / unpooled_height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, width);
    Dtype distval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, unpooled_height + pad_h);
        int wend = min(wstart + kernel_w, unpooled_width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        distval += bottom_data[ph * width + pw];
      }
    }
    top_data[index] = distval;
  }
}

template <typename Dtype>
void UnpoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  int count = bottom[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_set(top[0]->count(), Dtype(0.), top_data);
  // We'll get the mask from bottom[1] if it's of size >1.
  const bool use_bottom_mask = bottom.size() > 1;
  const Dtype* bottom_mask = NULL;
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnpoolingParameter_UnpoolMethod_MAX:
    if (use_bottom_mask) {
      bottom_mask = bottom[1]->gpu_data();
    } 
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxUnpoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,
        bottom_mask);
    break;
  case UnpoolingParameter_UnpoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AveUnpoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        top[0]->count(), bottom_data, bottom[0]->num(), channels_,
        unpooled_height_, unpooled_width_, height_, width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  case UnpoolingParameter_UnpoolMethod_TILE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    TileUnpoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        top[0]->count(), bottom_data, bottom[0]->num(), channels_,
        unpooled_height_, unpooled_width_, height_, width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxUnpoolBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* bottom_mask, const int num, const int channels,
    const int height, const int width, const int unpooled_height,
    const int unpooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int pw = index % width;
    int ph = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    int uph = max(0, min(ph * stride_h - pad_h, unpooled_height - 1));
    int upw = max(0, min(pw * stride_w - pad_w, unpooled_width - 1));
    int unpooled_index = uph * unpooled_width + upw;

    top_diff += (n * channels + c) * unpooled_height * unpooled_width;
    if (bottom_mask) {
      const int mask_index = bottom_mask[index];
      bottom_diff[index] = top_diff[mask_index]; 
    } else {
      bottom_diff[index] = top_diff[unpooled_index];
    } 
  }
}


template <typename Dtype>
__global__ void AveUnpoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int unpooled_height,
    const int unpooled_width, const int height, const int width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % width;
    int ph = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, unpooled_height + pad_h);
    int wend = min(wstart + kernel_w, unpooled_width + pad_w);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, unpooled_height);
    wend = min(wend, unpooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * unpooled_height * unpooled_width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        gradient += top_diff[h * unpooled_width + w];
      }
    }
    bottom_diff[index] = gradient / pool_size;
  }
}

template <typename Dtype>
__global__ void TileUnpoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int unpooled_height,
    const int unpooled_width, const int height, const int width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % width;
    int ph = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, unpooled_height + pad_h);
    int wend = min(wstart + kernel_w, unpooled_width + pad_w);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, unpooled_height);
    wend = min(wend, unpooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * unpooled_height * unpooled_width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        gradient += top_diff[h * unpooled_width + w];
      }
    }
    bottom_diff[index] = gradient / pool_size;
  }
}


template <typename Dtype>
void UnpoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll get the mask from bottom[1] if it's of size >1.
  const bool use_bottom_mask = bottom.size() > 1;
  const Dtype* bottom_mask = NULL;
  switch (this->layer_param_.unpooling_param().unpool()) {
  case UnpoolingParameter_UnpoolMethod_MAX:
    if (use_bottom_mask) {
      bottom_mask = bottom[1]->gpu_data();
    } 
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxUnpoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_mask, top[0]->num(), channels_,
        height_, width_, unpooled_height_, unpooled_width_,
        kernel_h_, kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_,
        bottom_diff);
    break;
  case UnpoolingParameter_UnpoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AveUnpoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), top_diff, top[0]->num(), channels_,
        unpooled_height_, unpooled_width_, height_, width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
    break;
  case UnpoolingParameter_UnpoolMethod_TILE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    TileUnpoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom[0]->count(), top_diff, top[0]->num(), channels_,
        unpooled_height_, unpooled_width_, height_, width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
    break;
  default:
    LOG(FATAL) << "Unknown unpooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(UnpoolingLayer);


}  // namespace caffe
