#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/center_projection_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

	template <typename Dtype>
	__global__ void L2Normalize(const int n, const Dtype* in, Dtype* out, int length){
		CUDA_KERNEL_LOOP(index, n) {
			Dtype sum = 0;
			for ( int i = 0; i < length; i++ )
			{
				sum = sum + in[ index*length + i ] * in[ index*length + i ];
			}
			sum = sqrt(sum)+1e-6;
			for ( int i = 0; i < length; i++ )
			{
				out[ index*length + i ] = in[ index*length + i ] / sum;
			}
		}
	}


	template <typename Dtype>
	void CenterProjectionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* bottom_data = bottom[ 0 ]->gpu_data();
		Dtype* top_data = top[ 0 ]->mutable_gpu_data();
		Dtype* weight_writable = this->blobs_[ 0 ]->mutable_gpu_data();
		const Dtype* weight = this->blobs_[ 0 ]->gpu_data();
		if ( M_ == 1 ) {
			caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, ( Dtype )1.,
				weight, bottom_data, ( Dtype )0., top_data);
			if ( bias_term_ )
				caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[ 0 ],
				this->blobs_[ 1 ]->gpu_data(), top_data);
		}
		else {
			// Step 1: Normalize weight
			L2Normalize<Dtype> << <CAFFE_GET_BLOCKS(N_), CAFFE_CUDA_NUM_THREADS >> >(N_, weight, weight_writable, K_);
			CUDA_POST_KERNEL_CHECK;
			/*Dtype* squared_data = squared_.mutable_gpu_data();
			caffe_gpu_powx(N_*K_, weight, Dtype(2), squared_data);
			Dtype normsqr;
			for (int i = 0; i<N_; ++i) {
			caffe_gpu_asum<Dtype>(K_, squared_data + i*K_, &normsqr);
			caffe_gpu_scale<Dtype>(K_, pow(normsqr, -0.5), weight + i*K_, weight_writable + i*K_);
			caffe_gpu_scale<Dtype>(K_, rescale_coeff_, weight + i*K_, weight_writable + i*K_);
			}*/
			// Step 2: Get projection
			caffe_gpu_gemm<Dtype>(CblasNoTrans,
				transpose_ ? CblasNoTrans : CblasTrans,
				M_, N_, K_, rescale_coeff_,
				bottom_data, weight, ( Dtype )0., top_data);
			if ( bias_term_ )
				caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, ( Dtype )1.,
				bias_multiplier_.gpu_data(),
				this->blobs_[ 1 ]->gpu_data(), ( Dtype )1., top_data);
		}
	}

	template <typename Dtype>
	void CenterProjectionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		if ( this->param_propagate_down_[ 0 ] ) {
			const Dtype* top_diff = top[ 0 ]->gpu_diff();
			const Dtype* bottom_data = bottom[ 0 ]->gpu_data();
			// Gradient with respect to weight
			if ( transpose_ ) {
				caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
					K_, N_, M_,
					rescale_coeff_, bottom_data, top_diff,
					( Dtype )1., this->blobs_[ 0 ]->mutable_gpu_diff());
			}
			else {
				caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
					N_, K_, M_,
					rescale_coeff_, top_diff, bottom_data,
					( Dtype )1., this->blobs_[ 0 ]->mutable_gpu_diff());
			}
		}
		if ( bias_term_ && this->param_propagate_down_[ 1 ] ) {
			const Dtype* top_diff = top[ 0 ]->gpu_diff();
			// Gradient with respect to bias
			caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, ( Dtype )1., top_diff,
				bias_multiplier_.gpu_data(), ( Dtype )1.,
				this->blobs_[ 1 ]->mutable_gpu_diff());
		}
		if ( propagate_down[ 0 ] ) {
			const Dtype* top_diff = top[ 0 ]->gpu_diff();
			// Gradient with respect to bottom data
			if ( transpose_ ) {
				caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
					M_, K_, N_,
					rescale_coeff_, top_diff, this->blobs_[0]->gpu_data(),
					( Dtype )0., bottom[ 0 ]->mutable_gpu_diff());
			}
			else {
				caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
					M_, K_, N_,
					rescale_coeff_, top_diff, this->blobs_[0]->gpu_data(),
					( Dtype )0., bottom[ 0 ]->mutable_gpu_diff());
			}
		}
	}

	INSTANTIATE_LAYER_GPU_FUNCS(CenterProjectionLayer);

}  // namespace caffe
